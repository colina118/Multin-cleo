
#include <hip/hip_runtime.h>
#include "stdio.h"
#include "stdlib.h"
#include "string.h"
#include "errno.h"


// Recibe dos matrices y las multiplica, la matriz c debe estar inicializada en 0
void matrixMult(unsigned long **a, unsigned long **b, unsigned long **c, int N)
{
    int i, j, k;
    for(i = 0; i < N; ++i)
    {
        for(j = 0; j < N; ++j)
        {
            for(k = 0; k < N; ++k)
            {
                c[i][j] += a[i][k] * b[k][j];
            }
        }
    }
}


int main(int argc, char * argv[])
{
    // Declaracion de variables
    int i, j, cont, N;
    unsigned long **a, **b, **c;
    char *p;
    hipEvent_t start, stop, startTotal, stopTotal;
    float tiempo, tiempoTotal;

    // Inicializar medidas de tiempo
    hipEventCreate(&start);
    hipEventCreate(&startTotal);
    hipEventCreate(&stop);
    hipEventCreate(&stopTotal);

    // Comenzar a medir tiempo total

    hipEventRecord(startTotal, 0);

    if(argc >= 2)
    {
        N = (int)strtol(argv[1], &p, 10);
        if (*p != '\0' && errno != 0)
        {
            printf("Primer parametro debe ser un numero.\n");
            printf("Error: %s\n", strerror(errno));
            return 1;
        }
    }
    else
    {
        printf("No hay valor de N de entrada, ingrese valor.\n");
        return 1;
    }

    // Reservar memoria para arreglo de arreglos
    a = (unsigned long**)malloc(N*sizeof(unsigned long*));
    b = (unsigned long**)malloc(N*sizeof(unsigned long*));
    c = (unsigned long**)malloc(N*sizeof(unsigned long*));

    // Reservar memoria para cada arreglo
    for(i = 0; i < N; ++i)
    {
        *(a+i) = (unsigned long*)malloc(N*sizeof(unsigned long));
        *(b+i) = (unsigned long*)malloc(N*sizeof(unsigned long));
        *(c+i) = (unsigned long*)malloc(N*sizeof(unsigned long));
    }


    // LLenar matrices con valores prueba
    for(i = 0; i < N; ++i)
    {
        for(j = 0; j < N; ++j)
        {
            a[i][j] = rand() % 1000 + 11;
            b[i][j] = rand() % 1000 + 11;
            c[i][j] = 0;
            ++cont;
        }
    }

    // Medir tiempo de calculo
    hipEventRecord(start, 0);

    // Hacer la multiplicacion
    matrixMult(a, b, c, N);

    // Finalizar tiempo de calculo
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    // Finalizar tiempo total
    hipEventRecord(stopTotal, 0);
    hipEventSynchronize(stopTotal);

    // calcular tiempos
    hipEventElapsedTime(&tiempo, start, stop);
    hipEventElapsedTime(&tiempoTotal, startTotal, stopTotal);

    // Imprimir que matrices se van a multiplicar
    if(argc >= 3 && !strcmp(argv[2], "2"))
    {
        printf("matriz a:\n");
        for(i = 0; i < N; ++i)
        {
            for(j = 0; j < N; ++j)
            {
                printf("%lu\t", a[i][j]);
            }
            printf("\n");
        }

        printf("\nmatriz b:\n");
        for(i = 0; i < N; ++i)
        {
            for(j = 0; j < N; ++j)
            {
                printf("%lu\t", b[i][j]);
            }
            printf("\n");
        }
    }

    // Imprimir resultado si el argumento es 1 o 2
    if(argc >= 3 && (!strcmp(argv[2], "2") || !strcmp(argv[2], "1")))
    {
        printf("\nmatriz resultado:\n");
        for(i = 0; i < N; ++i)
        {
            for(j = 0; j < N; ++j)
            {
                printf("%lu\t", c[i][j]);
            }
            printf("\n");
        }
    }


    printf("Tiempo total: %f ms.\n", tiempoTotal);
    printf("Tiempo de calculo: %f ms.\n", tiempo);

    // Liberar memoria
    for(i = 0; i < N; ++i)
    {
        free(*(a+i));
        free(*(b+i));
        free(*(c+i));
    }

    free(a);
    free(b);
    free(c);

    return 0;
}